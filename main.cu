#include<iostream>
#include <stdio.h>
#include<hip/hip_runtime.h>
#include<cmath>

using namespace std;

__global__ void whoami(void) {

    int block_id =
        blockIdx.x +
        blockIdx.y * gridDim.x +
        blockIdx.z * gridDim.x * gridDim.y;

    int block_offset =
        block_id *
        blockDim.x * blockDim.y * blockDim.z;

    int thread_offset =
        threadIdx.x +
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset;

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        id,
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
}


int main(int argc, char **argv){
    int b_x = 2, b_y = 3, b_z = 4;
    int t_x = 4, t_y = 4, t_z = 4;

    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    printf("%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x, b_y, b_z);
    dim3 threadsPerBlock(t_x, t_y, t_z);

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    hipDeviceSynchronize();
}